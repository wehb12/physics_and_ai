#include "hip/hip_runtime.h"
/******************************************************************************
Code: CUDA implementation of a sphere-sphere collision check
Implements:
Author:
Will Hinds      <w.hinds2@newcastle.ac.uk>
Description:

Carries out a sphere-sphere collision check on several spherical entities
then populates the data structures with the necessary info to compute a
collision manifold (of one point for a sphere-sphere collision)

*//////////////////////////////////////////////////////////////////////////////


// CUDA includes
#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>
#include <iostream>

#include "CUDA_SphereSphereCheck.cuh"

void CUDA_run(Vector3* positions, float* radii,
	Vector3* globalOnA, Vector3* globalOnB,
	Vector3* normal, float* penetration, int arrSize);
bool CUDA_init(int arrSize);
bool CUDA_free();

__global__
void CUDA_SphereSphereCheck(Vector3* cuda_pos, float* cuda_radius,
	Vector3* cuda_globalOnA, Vector3* cuda_globalOnB,
	Vector3* cuda_normal, float* cuda_penetration, int* cuda_arrSize)
{
	//printf("Radius [0] is %f\n", cuda_radius[0]);
	int start = blockIdx.x * blockDim.x + threadIdx.x;
	int jump = blockDim.x * gridDim.x;
	int collPairIndex = start;
	for (int i = start; i < *cuda_arrSize - 1; i += jump)
	{
		for (int j = i + 1; j < *cuda_arrSize; ++j)
		{
			Vector3 itoj = cuda_pos[i] - cuda_pos[j];
			float length = itoj.Length();
			//int collPairNum = index + j - 1;

			if (length < (cuda_radius[i] + cuda_radius[j]))		// collision detected
			{
				// i and j form a collison pair, from which we must construct a collision manifold
				cuda_globalOnA[collPairIndex] = (-itoj.Normalise() * cuda_radius[i]);
				cuda_globalOnB[collPairIndex] = (itoj.Normalise() * cuda_radius[j]);
				cuda_normal[collPairIndex] = itoj.Normalise();
				cuda_penetration[collPairIndex] = cuda_radius[i] + cuda_radius[j] - length;
			}
			else	// no collision detected
			{
				cuda_globalOnA[collPairIndex].ToZero();
				cuda_globalOnB[collPairIndex].ToZero();
				cuda_normal[collPairIndex].ToZero();
				cuda_penetration[collPairIndex] = 0;
			}
			++collPairIndex;
		}
		collPairIndex += jump - 1;
		//index += *cuda_arrSize - (i + 1);
	}
}

Vector3* cuda_pos = 0;
float* cuda_radius = 0;
Vector3* cuda_globalOnA = 0;
Vector3* cuda_globalOnB = 0;
Vector3* cuda_normal = 0;
float* cuda_penetration = 0;
int* cuda_arrSize = 0;

bool CUDA_init(int arrSize)
{
	hipError_t cudaStatus;
	bool success = true;
	// Error checking code from the default CUDA VS project
	//
	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		success = true;
	}

	// Allocate GPU buffers for all data
	cudaStatus = hipMalloc((void**)&cuda_pos, arrSize * sizeof(Vector3));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		success = true;
	}
	cudaStatus = hipMalloc((void**)&cuda_radius, arrSize * sizeof(float));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		success = true;
	}

	int maxCollPairs = arrSize * arrSize * 0.5;
	cudaStatus = hipMalloc((void**)&cuda_globalOnA, maxCollPairs * sizeof(Vector3));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		success = true;
	}
	cudaStatus = hipMalloc((void**)&cuda_globalOnB, maxCollPairs * sizeof(Vector3));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		success = true;
	}
	cudaStatus = hipMalloc((void**)&cuda_normal, maxCollPairs * sizeof(Vector3));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		success = true;
	}
	cudaStatus = hipMalloc((void**)&cuda_penetration, maxCollPairs * sizeof(float));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		success = true;
	}
	cudaStatus = hipMalloc((void**)&cuda_arrSize, sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		success = true;
	}

	return success;
}

bool CUDA_free()
{
	hipError_t cudaStatus;

	hipFree(cuda_pos);
	hipFree(cuda_radius);
	hipFree(cuda_globalOnA);
	hipFree(cuda_globalOnB);
	hipFree(cuda_normal);
	hipFree(cuda_penetration);
	hipFree(cuda_arrSize);

	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipDeviceReset failed!");
		return false;
	}

	return true;
}

void CUDA_run(Vector3* positions, float* radii,
	Vector3* globalOnA, Vector3* globalOnB,
	Vector3* normal, float* penetration, int arrSize)
{
	hipError_t cudaStatus;
	bool error = false;

	// copy data from host to GPU
	cudaStatus = hipMemcpy(cuda_pos, positions, arrSize * sizeof(Vector3), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		error = true;
	}
	cudaStatus = hipMemcpy(cuda_radius, radii, arrSize * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		error = true;
	}
	cudaStatus = hipMemcpy(cuda_arrSize, &arrSize, sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		error = true;
	}

	int blockSize = 256;
	// calculate gridSize from https://devblogs.nvidia.com/parallelforall/even-easier-introduction-cuda/
	int gridSize = (arrSize + blockSize - 1) / blockSize;
	if (!error)
		CUDA_SphereSphereCheck<<<gridSize, blockSize>>>(cuda_pos, cuda_radius, cuda_globalOnA, cuda_globalOnB, cuda_normal, cuda_penetration, cuda_arrSize);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);

	int maxCollPairs = arrSize * arrSize * 0.5;
	// copy outputs back to host memory
	cudaStatus = hipMemcpy(globalOnA, cuda_globalOnA, maxCollPairs * sizeof(Vector3), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "hipMalloc failed!");
	cudaStatus = hipMemcpy(globalOnB, cuda_globalOnB, maxCollPairs * sizeof(Vector3), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "hipMalloc failed!");
	cudaStatus = hipMemcpy(normal, cuda_normal, maxCollPairs * sizeof(Vector3), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "hipMalloc failed!");
	cudaStatus = hipMemcpy(penetration, cuda_penetration, maxCollPairs * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "hipMalloc failed!");

	return;
}


//#include "hip/hip_runtime.h"
//#include ""
//#include "CUDA_SphereSphereCheck.cuh"
//
//#include <stdio.h>
//
//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
//
//__global__ void addKernel(int *c, const int *a, const int *b)
//{
//	int i = threadIdx.x;
//	c[i] = a[i] + b[i];
//}
//
//int CUDA_run()
//{
//	const int arraySize = 5;
//	const int a[arraySize] = { 1, 2, 3, 4, 5 };
//	const int b[arraySize] = { 10, 20, 30, 40, 50 };
//	int c[arraySize] = { 0 };
//
//	// Add vectors in parallel.
//	hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "addWithCuda failed!");
//		return 1;
//	}
//
//	printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
//		c[0], c[1], c[2], c[3], c[4]);
//
//	// hipDeviceReset must be called before exiting in order for profiling and
//	// tracing tools such as Nsight and Visual Profiler to show complete traces.
//	cudaStatus = hipDeviceReset();
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipDeviceReset failed!");
//		return 1;
//	}
//
//	return 0;
//}
//
//// Helper function for using CUDA to add vectors in parallel.
//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
//{
//	int *dev_a = 0;
//	int *dev_b = 0;
//	int *dev_c = 0;
//	hipError_t cudaStatus;
//
//	// Choose which GPU to run on, change this on a multi-GPU system.
//	cudaStatus = hipSetDevice(0);
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
//		goto Error;
//	}
//
//	// Allocate GPU buffers for three vectors (two input, one output)    .
//	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipMalloc failed!");
//		goto Error;
//	}
//
//	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipMalloc failed!");
//		goto Error;
//	}
//
//	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipMalloc failed!");
//		goto Error;
//	}
//
//	// Copy input vectors from host memory to GPU buffers.
//	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipMemcpy failed!");
//		goto Error;
//	}
//
//	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipMemcpy failed!");
//		goto Error;
//	}
//
//	// Launch a kernel on the GPU with one thread for each element.
//	addKernel <<<1, size >>>(dev_c, dev_a, dev_b);
//
//	// Check for any errors launching the kernel
//	cudaStatus = hipGetLastError();
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
//		goto Error;
//	}
//
//	// hipDeviceSynchronize waits for the kernel to finish, and returns
//	// any errors encountered during the launch.
//	cudaStatus = hipDeviceSynchronize();
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//		goto Error;
//	}
//
//	// Copy output vector from GPU buffer to host memory.
//	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipMemcpy failed!");
//		goto Error;
//	}
//
//Error:
//	hipFree(dev_c);
//	hipFree(dev_a);
//	hipFree(dev_b);
//
//	return cudaStatus;
//}
