#include "hip/hip_runtime.h"
#pragma once

// CUDA includes
#include<hip/hip_runtime.h>
#include<hip/hip_vector_types.h>

#include "../nclgl/Vector3.h"

__global__
void CUDA_SphereSphereCheck(Vector3* cu_pos, float* cu_radius,
	Vector3* cu_globalOnA, Vector3* cu_globalOnB,
	Vector3* cu_normal, float* cu_penetration, int entities)
{
	for (int i = 0; i < entities - 1; ++i)
	{
		for (int j = i + 1; j < entities; ++j)
		{
			Vector3 itoj = cu_pos[i] - cu_pos[j];
			float length = itoj.Length();

			if (length < (cu_radius[i] + cu_radius[j]))
			{

			}
		}
	}
}

void CUDA_run(Vector3* cu_pos, float* cu_radius,
	Vector3* cu_globalOnA, Vector3* cu_globalOnB,
	Vector3* cu_normal, float* cu_penetration, int entities)
{
	CUDA_SphereSphereCheck<<<1, 1>>>(positions, radii, globalOnA, globalOnB, normal, penetration, arrSize);
}
