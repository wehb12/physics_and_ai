#include "hip/hip_runtime.h"
/******************************************************************************
Code: CUDA implementation of a sphere-sphere collision check
Implements:
Author:
Will Hinds      <w.hinds2@newcastle.ac.uk>
Description:

Carries out a sphere-sphere collision check on several spherical entities
then populates the data structures with the necessary info to compute a
collision manifold (of one point for a sphere-sphere collision)

*//////////////////////////////////////////////////////////////////////////////

#pragma once

// CUDA includes
#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>
#include <iostream>

#include "CUDA_SphereSphereCheck.cuh"

void CUDA_run(Vector3* positions, float* radii,
	Vector3* globalOnA, Vector3* globalOnB,
	Vector3* normal, float* penetration, int arrSize);

__global__
void CUDA_SphereSphereCheck(Vector3* cuda_pos, float* cuda_radius,
	Vector3* cuda_globalOnA, Vector3* cuda_globalOnB,
	Vector3* cuda_normal, float* cuda_penetration, int* cuda_arrSize)
{
	for (int i = 0; i < *cuda_arrSize - 1; ++i)
	{
		for (int j = i + 1; j < *cuda_arrSize; ++j)
		{
			Vector3 itoj = cuda_pos[i] - cuda_pos[j];
			float length = itoj.Length();

			if (length < (cuda_radius[i] + cuda_radius[j]))
			{

			}
		}
	}
}

void CUDA_run(Vector3* positions, float* radii,
	Vector3* globalOnA, Vector3* globalOnB,
	Vector3* normal, float* penetration, int arrSize)
{
	Vector3* cuda_pos = 0;
	float* cuda_radius = 0;
	Vector3* cuda_globalOnA = 0;
	Vector3* cuda_globalOnB = 0;
	Vector3* cuda_normal = 0;
	float* cuda_penetration = 0;
	int* cuda_arrSize = 0;
	hipError_t cudaStatus;
	bool error = false;

	// Error checking code from the default CUDA VS project
	//
	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		error = true;
	}

	// Allocate GPU buffers for all data
	cudaStatus = hipMalloc((void**)&cuda_pos, arrSize * sizeof(Vector3));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		error = true;
	}
	cudaStatus = hipMalloc((void**)&cuda_radius, arrSize * sizeof(float));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		error = true;
	}
	cudaStatus = hipMalloc((void**)&cuda_globalOnA, arrSize * sizeof(Vector3));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		error = true;
	}
	cudaStatus = hipMalloc((void**)&cuda_globalOnB, arrSize * sizeof(Vector3));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		error = true;
	}
	cudaStatus = hipMalloc((void**)&cuda_normal, arrSize * sizeof(Vector3));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		error = true;
	}
	cudaStatus = hipMalloc((void**)&cuda_penetration, arrSize * sizeof(float));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		error = true;
	}
	cudaStatus = hipMalloc((void**)&cuda_arrSize, sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		error = true;
	}

	// copy data from host to GPU
	cudaStatus = hipMemcpy(cuda_pos, positions, arrSize * sizeof(Vector3), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		error = true;
	}
	cudaStatus = hipMemcpy(cuda_radius, radii, arrSize * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		error = true;
	}

	if (!error)
		CUDA_SphereSphereCheck<<<1, 1>>>(cuda_pos, cuda_radius, cuda_globalOnA, cuda_globalOnB, cuda_normal, cuda_penetration, cuda_arrSize);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);

	// copy outputs back to host memory
	cudaStatus = hipMemcpy(globalOnA, cuda_globalOnA, arrSize * sizeof(Vector3), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "hipMalloc failed!");
	cudaStatus = hipMemcpy(globalOnB, cuda_globalOnB, arrSize * sizeof(Vector3), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "hipMalloc failed!");
	cudaStatus = hipMemcpy(normal, cuda_normal, arrSize * sizeof(Vector3), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "hipMalloc failed!");
	cudaStatus = hipMemcpy(penetration, cuda_penetration, arrSize * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "hipMalloc failed!");

	hipFree(cuda_pos);
	hipFree(cuda_radius);
	hipFree(cuda_globalOnA);
	hipFree(cuda_globalOnB);
	hipFree(cuda_normal);
	hipFree(cuda_penetration);
	hipFree(cuda_arrSize);

	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "hipDeviceReset failed!");

	return;
}


//#include "hip/hip_runtime.h"
//#include ""
//#include "CUDA_SphereSphereCheck.cuh"
//
//#include <stdio.h>
//
//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
//
//__global__ void addKernel(int *c, const int *a, const int *b)
//{
//	int i = threadIdx.x;
//	c[i] = a[i] + b[i];
//}
//
//int CUDA_run()
//{
//	const int arraySize = 5;
//	const int a[arraySize] = { 1, 2, 3, 4, 5 };
//	const int b[arraySize] = { 10, 20, 30, 40, 50 };
//	int c[arraySize] = { 0 };
//
//	// Add vectors in parallel.
//	hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "addWithCuda failed!");
//		return 1;
//	}
//
//	printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
//		c[0], c[1], c[2], c[3], c[4]);
//
//	// hipDeviceReset must be called before exiting in order for profiling and
//	// tracing tools such as Nsight and Visual Profiler to show complete traces.
//	cudaStatus = hipDeviceReset();
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipDeviceReset failed!");
//		return 1;
//	}
//
//	return 0;
//}
//
//// Helper function for using CUDA to add vectors in parallel.
//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
//{
//	int *dev_a = 0;
//	int *dev_b = 0;
//	int *dev_c = 0;
//	hipError_t cudaStatus;
//
//	// Choose which GPU to run on, change this on a multi-GPU system.
//	cudaStatus = hipSetDevice(0);
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
//		goto Error;
//	}
//
//	// Allocate GPU buffers for three vectors (two input, one output)    .
//	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipMalloc failed!");
//		goto Error;
//	}
//
//	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipMalloc failed!");
//		goto Error;
//	}
//
//	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipMalloc failed!");
//		goto Error;
//	}
//
//	// Copy input vectors from host memory to GPU buffers.
//	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipMemcpy failed!");
//		goto Error;
//	}
//
//	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipMemcpy failed!");
//		goto Error;
//	}
//
//	// Launch a kernel on the GPU with one thread for each element.
//	addKernel <<<1, size >>>(dev_c, dev_a, dev_b);
//
//	// Check for any errors launching the kernel
//	cudaStatus = hipGetLastError();
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
//		goto Error;
//	}
//
//	// hipDeviceSynchronize waits for the kernel to finish, and returns
//	// any errors encountered during the launch.
//	cudaStatus = hipDeviceSynchronize();
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//		goto Error;
//	}
//
//	// Copy output vector from GPU buffer to host memory.
//	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipMemcpy failed!");
//		goto Error;
//	}
//
//Error:
//	hipFree(dev_c);
//	hipFree(dev_a);
//	hipFree(dev_b);
//
//	return cudaStatus;
//}
