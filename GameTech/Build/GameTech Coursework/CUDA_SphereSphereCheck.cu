#include "hip/hip_runtime.h"
/******************************************************************************
Code: CUDA implementation of a sphere-sphere collision check
Implements:
Author:
Will Hinds      <w.hinds2@newcastle.ac.uk>
Description:

Carries out a sphere-sphere collision check on several spherical entities
then populates the data structures with the necessary info to compute a
collision manifold (of one point for a sphere-sphere collision)

*//////////////////////////////////////////////////////////////////////////////


// CUDA includes
#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>
#include <iostream>

#include "../nclgl/Vector3.h"

#include "CUDA_SphereSphereCheck.cuh"

void CUDA_run(Vector3* positions, float* radii,
	Vector3* globalOnA, Vector3* globalOnB,
	Vector3* normal, float* penetration, int* cuda_nodeAIndex,
	int* cuda_nodeBIndex, int arrSize);
bool CUDA_init(int arrSize);
bool CUDA_free();

__global__
void CUDA_SphereSphereCheck(Vector3* cuda_pos, float* cuda_radius,
	Vector3* cuda_globalOnA, Vector3* cuda_globalOnB,
	Vector3* cuda_normal, float* cuda_penetration, int* cuda_nodeAIndex, int* cuda_nodeBIndex, int* cuda_arrSize)
{
	int start = blockIdx.x * blockDim.x + threadIdx.x;
	int collPairIndex = 0;
	for (int i = 1; i <= start; ++i)
	{
		collPairIndex += (*cuda_arrSize - i);
	}
	for (int j = start + 1; j < *cuda_arrSize; ++j)
	{
		Vector3 itoj = cuda_pos[start] - cuda_pos[j];
		float length = itoj.Length();

		if (length < (cuda_radius[start] + cuda_radius[j]))		// collision detected
		{
			// i and j form a collison pair, from which we must construct a collision manifold
			cuda_globalOnA[collPairIndex] = cuda_pos[start] - (itoj.Normalise() * cuda_radius[start]);
			cuda_globalOnB[collPairIndex] = cuda_pos[j] + (itoj.Normalise() * cuda_radius[j]);
			cuda_normal[collPairIndex] = -itoj.Normalise();
			cuda_penetration[collPairIndex] = length - (cuda_radius[start] + cuda_radius[j]);
			cuda_nodeAIndex[collPairIndex] = start;
			cuda_nodeBIndex[collPairIndex] = j;
		}
		else	// no collision detected
		{
			cuda_globalOnA[collPairIndex].ToZero();
			cuda_globalOnB[collPairIndex].ToZero();
			cuda_normal[collPairIndex].ToZero();
			cuda_penetration[collPairIndex] = 0;
			cuda_nodeAIndex[collPairIndex] = -1;
			cuda_nodeBIndex[collPairIndex] = -1;
		}
		++collPairIndex;
	}
}

Vector3* cuda_pos = 0;
float* cuda_radius = 0;
Vector3* cuda_globalOnA = 0;
Vector3* cuda_globalOnB = 0;
Vector3* cuda_normal = 0;
float* cuda_penetration = 0;
int* cuda_arrSize = 0;
int* cuda_nodeAIndex = 0;
int* cuda_nodeBIndex = 0;

bool CUDA_init(int arrSize)
{
	hipError_t cudaStatus;
	bool success = true;
	// Error checking code from the default CUDA VS project
	//
	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		success = true;
	}

	// Allocate GPU buffers for all data
	cudaStatus = hipMalloc((void**)&cuda_pos, arrSize * sizeof(Vector3));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		success = true;
	}
	cudaStatus = hipMalloc((void**)&cuda_radius, arrSize * sizeof(float));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		success = true;
	}

	int maxCollPairs = arrSize * arrSize * 0.5;
	cudaStatus = hipMalloc((void**)&cuda_globalOnA, maxCollPairs * sizeof(Vector3));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		success = true;
	}
	cudaStatus = hipMalloc((void**)&cuda_globalOnB, maxCollPairs * sizeof(Vector3));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		success = true;
	}
	cudaStatus = hipMalloc((void**)&cuda_normal, maxCollPairs * sizeof(Vector3));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		success = true;
	}
	cudaStatus = hipMalloc((void**)&cuda_penetration, maxCollPairs * sizeof(float));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		success = true;
	}
	cudaStatus = hipMalloc((void**)&cuda_nodeAIndex, maxCollPairs * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		success = true;
	}
	cudaStatus = hipMalloc((void**)&cuda_nodeBIndex, maxCollPairs * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		success = true;
	}
	cudaStatus = hipMalloc((void**)&cuda_arrSize, sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		success = true;
	}

	return success;
}

bool CUDA_free()
{
	hipError_t cudaStatus;

	hipFree(cuda_pos);
	hipFree(cuda_radius);
	hipFree(cuda_globalOnA);
	hipFree(cuda_globalOnB);
	hipFree(cuda_normal);
	hipFree(cuda_penetration);
	hipFree(cuda_nodeAIndex);
	hipFree(cuda_nodeBIndex);
	hipFree(cuda_arrSize);

	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipDeviceReset failed!");
		return false;
	}

	return true;
}

void CUDA_run(Vector3* positions, float* radii,
	Vector3* globalOnA, Vector3* globalOnB,
	Vector3* normal, float* penetration, int* indexA,
	int* indexB, int arrSize)
{
	hipError_t cudaStatus;
	bool error = false;

	// copy data from host to GPU
	cudaStatus = hipMemcpy(cuda_pos, positions, arrSize * sizeof(Vector3), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		error = true;
	}
	cudaStatus = hipMemcpy(cuda_radius, radii, arrSize * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		error = true;
	}
	cudaStatus = hipMemcpy(cuda_arrSize, &arrSize, sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		error = true;
	}

	int blockSize = 256;
	// calculate gridSize from https://devblogs.nvidia.com/parallelforall/even-easier-introduction-cuda/
	int gridSize = (arrSize + blockSize - 1) / blockSize;
	if (!error)
		CUDA_SphereSphereCheck << <gridSize, 600 >> >(cuda_pos, cuda_radius, cuda_globalOnA, cuda_globalOnB, cuda_normal, cuda_penetration, cuda_nodeAIndex, cuda_nodeBIndex, cuda_arrSize);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);

	int maxCollPairs = arrSize * arrSize * 0.5;
	// copy outputs back to host memory
	cudaStatus = hipMemcpy(globalOnA, cuda_globalOnA, maxCollPairs * sizeof(Vector3), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "hipMalloc failed!");
	cudaStatus = hipMemcpy(globalOnB, cuda_globalOnB, maxCollPairs * sizeof(Vector3), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "hipMalloc failed!");
	cudaStatus = hipMemcpy(normal, cuda_normal, maxCollPairs * sizeof(Vector3), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "hipMalloc failed!");
	cudaStatus = hipMemcpy(penetration, cuda_penetration, maxCollPairs * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "hipMalloc failed!");
	cudaStatus = hipMemcpy(indexA, cuda_nodeAIndex, maxCollPairs * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "hipMalloc failed!");
	cudaStatus = hipMemcpy(indexB, cuda_nodeBIndex, maxCollPairs * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "hipMalloc failed!");

	return;
}
