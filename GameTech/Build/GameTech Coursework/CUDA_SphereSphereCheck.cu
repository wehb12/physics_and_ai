#include "hip/hip_runtime.h"
/******************************************************************************
Code: CUDA implementation of a sphere-sphere collision check
Implements:
Author:
Will Hinds      <w.hinds2@newcastle.ac.uk>
Description:

Carries out a sphere-sphere collision check on several spherical entities
then populates the data structures with the necessary info to compute a
collision manifold (of one point for a sphere-sphere collision)

*//////////////////////////////////////////////////////////////////////////////


// CUDA includes
#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>
#include <iostream>

#include "../nclgl/Vector3.h"

#include "CUDA_SphereSphereCheck.cuh"

void CUDA_run(Vector3* positions, float* radii,
	Vector3* velocityIn, Vector3* velocityOut, int arrSize);
bool CUDA_init(int arrSize);
bool CUDA_free();

//__global__
//void CUDA_SphereSphereCheck(Vector3* cuda_pos, float* cuda_radius,
//	Vector3* cuda_globalOnA, Vector3* cuda_globalOnB,
//	Vector3* cuda_normal, float* cuda_penetration, int* cuda_nodeAIndex, int* cuda_nodeBIndex, int* cuda_arrSize)
//{
//	int start = blockIdx.x * blockDim.x + threadIdx.x;
//	int collPairIndex = 0;
//	for (int i = 1; i <= start; ++i)
//	{
//		collPairIndex += (*cuda_arrSize - i);
//	}
//	for (int j = start + 1; j < *cuda_arrSize; ++j)
//	{
//		Vector3 itoj = cuda_pos[start] - cuda_pos[j];
//		float length = itoj.Length();
//
//		if (length < (cuda_radius[start] + cuda_radius[j]))		// collision detected
//		{
//			// i and j form a collison pair, from which we must construct a collision manifold
//			cuda_globalOnA[collPairIndex] = cuda_pos[start] - (itoj.Normalise() * cuda_radius[start]);
//			cuda_globalOnB[collPairIndex] = cuda_pos[j] + (itoj.Normalise() * cuda_radius[j]);
//			cuda_normal[collPairIndex] = -itoj.Normalise();
//			cuda_penetration[collPairIndex] = length - (cuda_radius[start] + cuda_radius[j]);
//			cuda_nodeAIndex[collPairIndex] = start;
//			cuda_nodeBIndex[collPairIndex] = j;
//		}
//		else	// no collision detected
//		{
//			cuda_globalOnA[collPairIndex].ToZero();
//			cuda_globalOnB[collPairIndex].ToZero();
//			cuda_normal[collPairIndex].ToZero();
//			cuda_penetration[collPairIndex] = 0;
//			cuda_nodeAIndex[collPairIndex] = -1;
//			cuda_nodeBIndex[collPairIndex] = -1;
//		}
//		++collPairIndex;
//	}
//}

__global__
void CUDA_SphereSphereCheck(Vector3* cuda_pos, float* cuda_radius,
	Vector3* cuda_velocity, Vector3* cuda_velocityOut, int* cuda_arrSize)
{
	int start = blockIdx.x * blockDim.x + threadIdx.x;
	int collPairIndex = 0;
	for (int i = 1; i <= start; ++i)
	{
		collPairIndex += (*cuda_arrSize - i);
	}
	for (int j = start + 1; j < *cuda_arrSize; ++j)
	{
		Vector3 itoj = cuda_pos[j] - cuda_pos[start];
		float length = itoj.Length();

		if (length < (cuda_radius[start] + cuda_radius[j]))		// collision detected
		{
			// start and j form a collison pair
			Vector3 itojN = itoj / length;

			float itojNVel = Vector3::Dot(cuda_velocity[j] - cuda_velocity[start], itojN);
			float jn = -(itojNVel * (1.5f));

			float overlap = (cuda_radius[start] + cuda_radius[j]) - length;
			float b = overlap * 0.05f * 60.0f;

			jn += b;

			jn = jn > 0.0f ? jn : 0.0f;

			cuda_velocityOut[start] -= itojN * (jn * 0.5f);
			cuda_velocityOut[j] += itojN * (jn * 0.5f);
		}
		else	// no collision detected
		{
			cuda_globalOnA[collPairIndex].ToZero();
			cuda_globalOnB[collPairIndex].ToZero();
			cuda_normal[collPairIndex].ToZero();
			cuda_penetration[collPairIndex] = 0;
			cuda_nodeAIndex[collPairIndex] = -1;
			cuda_nodeBIndex[collPairIndex] = -1;
		}
		++collPairIndex;
	}
}

Vector3* cuda_pos = 0;
float* cuda_radius = 0;
Vector3* cuda_velocity = 0;
Vector3* cuda_velocityOut = 0;
int* cuda_arrSize = 0;

bool CUDA_init(int arrSize)
{
	hipError_t cudaStatus;
	bool success = true;
	// Error checking code from the default CUDA VS project
	//
	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		success = true;
	}

	// Allocate GPU buffers for all data
	cudaStatus = hipMalloc((void**)&cuda_pos, arrSize * sizeof(Vector3));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		success = true;
	}
	cudaStatus = hipMalloc((void**)&cuda_radius, arrSize * sizeof(float));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		success = true;
	}

	int maxCollPairs = arrSize * arrSize * 0.5;
	cudaStatus = hipMalloc((void**)&cuda_velocity, arrSize * sizeof(Vector3));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		success = true;
	}
	cudaStatus = hipMalloc((void**)&cuda_velocityOut, maxCollPairs * sizeof(Vector3));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		success = true;
	}
	cudaStatus = hipMalloc((void**)&cuda_arrSize, sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		success = true;
	}

	return success;
}

bool CUDA_free()
{
	hipError_t cudaStatus;

	hipFree(cuda_pos);
	hipFree(cuda_radius);
	hipFree(cuda_velocity);
	hipFree(cuda_velocityOut);
	hipFree(cuda_arrSize);

	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipDeviceReset failed!");
		return false;
	}

	return true;
}

void CUDA_run(Vector3* positions, float* radii,
	Vector3* velocityIn, Vector3* velocityOut, int arrSize)
{
	hipError_t cudaStatus;
	bool error = false;

	// copy data from host to GPU
	cudaStatus = hipMemcpy(cuda_pos, positions, arrSize * sizeof(Vector3), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		error = true;
	}
	cudaStatus = hipMemcpy(cuda_radius, radii, arrSize * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		error = true;
	}
	cudaStatus = hipMemcpy(cuda_arrSize, &arrSize, sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		error = true;
	}
	cudaStatus = hipMemcpy(cuda_velocity, velocityIn, arrSize * sizeof(Vector3), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		error = true;
	}
	int maxCollPairs = arrSize * arrSize * 0.5;
	cudaStatus = hipMemcpy(cuda_velocityOut, velocityIn, maxCollPairs * sizeof(Vector3), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		error = true;
	}

	int blockSize = 256;
	// calculate gridSize from https://devblogs.nvidia.com/parallelforall/even-easier-introduction-cuda/
	int gridSize = (arrSize + blockSize - 1) / blockSize;
	if (!error)
		CUDA_SphereSphereCheck << <gridSize, 600 >> >(cuda_pos, cuda_radius, cuda_velocity, cuda_velocityOut, cuda_arrSize);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);

	// copy outputs back to host memory
	cudaStatus = hipMemcpy(velocityOut, cuda_velocityOut, maxCollPairs * sizeof(Vector3), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "hipMalloc failed!");
	//cudaStatus = hipMemcpy(normal, cuda_normal, maxCollPairs * sizeof(Vector3), hipMemcpyDeviceToHost);
	//if (cudaStatus != hipSuccess)
	//	fprintf(stderr, "hipMalloc failed!");
	//cudaStatus = hipMemcpy(penetration, cuda_penetration, maxCollPairs * sizeof(float), hipMemcpyDeviceToHost);
	//if (cudaStatus != hipSuccess)
	//	fprintf(stderr, "hipMalloc failed!");
	//cudaStatus = hipMemcpy(indexA, cuda_nodeAIndex, maxCollPairs * sizeof(int), hipMemcpyDeviceToHost);
	//if (cudaStatus != hipSuccess)
	//	fprintf(stderr, "hipMalloc failed!");
	//cudaStatus = hipMemcpy(indexB, cuda_nodeBIndex, maxCollPairs * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "hipMalloc failed!");

	return;
}
